#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdint>
#include <cmath>
#include <bitset>
#include <string>
#include <filesystem>
#include "stb_image.h"

using namespace std;
using namespace chrono;
namespace fs = std::filesystem;

#define RESIZE_W 32
#define RESIZE_H 32
#define DCT_SIZE 32
#define HASH_SIZE 8
#define MAX_STREAMS 8

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__device__ float dct_coeff(int k, int N) {
    return (k == 0) ? sqrtf(1.0f / N) : sqrtf(2.0f / N);
}

__global__ void ucharToGrayscaleKernel(const unsigned char* input, float* output, int size, int channels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    if (channels == 3) {
        float r = input[i * 3 + 0];
        float g = input[i * 3 + 1];
        float b = input[i * 3 + 2];
        output[i] = 0.299f * r + 0.587f * g + 0.114f * b;
    } else {
        output[i] = input[i];
    }
}

__global__ void resizeBilinearKernel(const float* input, float* output, int inW, int inH, int outW, int outH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= outW || y >= outH) return;

    float scaleX = static_cast<float>(inW) / outW;
    float scaleY = static_cast<float>(inH) / outH;

    float srcX = (x + 0.5f) * scaleX - 0.5f;
    float srcY = (y + 0.5f) * scaleY - 0.5f;

    int x0 = max(0, min(inW - 2, static_cast<int>(floorf(srcX))));
    int y0 = max(0, min(inH - 2, static_cast<int>(floorf(srcY))));

    float dx = srcX - x0;
    float dy = srcY - y0;

    float val =
        (1 - dx) * (1 - dy) * input[y0 * inW + x0] +
        dx * (1 - dy) * input[y0 * inW + (x0 + 1)] +
        (1 - dx) * dy * input[(y0 + 1) * inW + x0] +
        dx * dy * input[(y0 + 1) * inW + (x0 + 1)];

    output[y * outW + x] = val;
}

__global__ void dctRowsKernel(const float* input, float* output, int N) {
    int row = blockIdx.x;
    int u = threadIdx.x;

    if (row < N && u < N) {
        float sum = 0.0f;
        for (int x = 0; x < N; ++x) {
            sum += input[row * N + x] * cosf((M_PI * (2 * x + 1) * u) / (2.0f * N));
        }
        output[row * N + u] = dct_coeff(u, N) * sum;
    }
}

__global__ void dctColsKernel(const float* input, float* output, int N) {
    int col = blockIdx.x;
    int v = threadIdx.x;

    if (col < N && v < N) {
        float sum = 0.0f;
        for (int y = 0; y < N; ++y) {
            sum += input[y * N + col] * cosf((M_PI * (2 * y + 1) * v) / (2.0f * N));
        }
        output[v * N + col] = dct_coeff(v, N) * sum;
    }
}

__global__ void generateHashKernel(const float* dct, uint64_t* hashOut) {
    __shared__ float block[HASH_SIZE * HASH_SIZE];
    int idx = threadIdx.x;
    if (idx < HASH_SIZE * HASH_SIZE) {
        int i = idx / HASH_SIZE;
        int j = idx % HASH_SIZE;
        block[idx] = dct[i * DCT_SIZE + j];
    }
    __syncthreads();

    if (idx == 0) {
        float sum = 0;
        for (int i = 1; i < HASH_SIZE * HASH_SIZE; ++i) // skip DC
            sum += block[i];
        float avg = sum / (HASH_SIZE * HASH_SIZE - 1);

        uint64_t hash = 0;
        for (int i = 1; i < HASH_SIZE * HASH_SIZE; ++i)
            if (block[i] > avg)
                hash |= (1ULL << (i - 1));
        *hashOut = hash;
    }
}

void computeHashAsync(unsigned char* img, int width, int height, int channels, hipStream_t stream, uint64_t* h_hash) {
    int imgSize = width * height;
    size_t imgBytes = imgSize * channels * sizeof(unsigned char);
    size_t grayBytes = imgSize * sizeof(float);
    size_t resizedBytes = RESIZE_W * RESIZE_H * sizeof(float);

    unsigned char* d_img;
    float *d_gray, *d_resized, *d_tempDCT, *d_finalDCT;
    uint64_t* d_hash;

    CHECK_CUDA(hipMallocAsync(&d_img, imgBytes, stream));
    CHECK_CUDA(hipMallocAsync(&d_gray, grayBytes, stream));
    CHECK_CUDA(hipMallocAsync(&d_resized, resizedBytes, stream));
    CHECK_CUDA(hipMallocAsync(&d_tempDCT, resizedBytes, stream));
    CHECK_CUDA(hipMallocAsync(&d_finalDCT, resizedBytes, stream));
    CHECK_CUDA(hipMallocAsync(&d_hash, sizeof(uint64_t), stream));

    CHECK_CUDA(hipMemcpyAsync(d_img, img, imgBytes, hipMemcpyHostToDevice, stream));

    int threads = 256;
    int blocks = (imgSize + threads - 1) / threads;
    ucharToGrayscaleKernel<<<blocks, threads, 0, stream>>>(d_img, d_gray, imgSize, channels);

    dim3 t2D(16, 16);
    dim3 b2D((RESIZE_W + 15) / 16, (RESIZE_H + 15) / 16);
    resizeBilinearKernel<<<b2D, t2D, 0, stream>>>(d_gray, d_resized, width, height, RESIZE_W, RESIZE_H);

    dctRowsKernel<<<RESIZE_H, RESIZE_W, 0, stream>>>(d_resized, d_tempDCT, RESIZE_W);
    dctColsKernel<<<RESIZE_W, RESIZE_H, 0, stream>>>(d_tempDCT, d_finalDCT, RESIZE_W);

    generateHashKernel<<<1, HASH_SIZE * HASH_SIZE, 0, stream>>>(d_finalDCT, d_hash);
    CHECK_CUDA(hipMemcpyAsync(h_hash, d_hash, sizeof(uint64_t), hipMemcpyDeviceToHost, stream));

    CHECK_CUDA(hipFreeAsync(d_img, stream));
    CHECK_CUDA(hipFreeAsync(d_gray, stream));
    CHECK_CUDA(hipFreeAsync(d_resized, stream));
    CHECK_CUDA(hipFreeAsync(d_tempDCT, stream));
    CHECK_CUDA(hipFreeAsync(d_finalDCT, stream));
    CHECK_CUDA(hipFreeAsync(d_hash, stream));
}

uint64_t computeHashGPU(unsigned char* img, int width, int height, int channels) {
    int imgSize = width * height;

    float* d_grayInput;
    float* d_resized;
    float* d_dctTemp;
    float* d_dctFinal;
    uint64_t* d_hash;

    CHECK_CUDA(hipMalloc(&d_grayInput, imgSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_resized, RESIZE_W * RESIZE_H * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dctTemp, RESIZE_W * RESIZE_H * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dctFinal, RESIZE_W * RESIZE_H * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_hash, sizeof(uint64_t)));

    unsigned char* d_img;
    CHECK_CUDA(hipMalloc(&d_img, imgSize * channels * sizeof(unsigned char)));
    CHECK_CUDA(hipMemcpy(d_img, img, imgSize * channels * sizeof(unsigned char), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (imgSize + threads - 1) / threads;
    ucharToGrayscaleKernel<<<blocks, threads>>>(d_img, d_grayInput, imgSize, channels);

    dim3 threads2D(16, 16);
    dim3 blocks2D((RESIZE_W + 15) / 16, (RESIZE_H + 15) / 16);
    resizeBilinearKernel<<<blocks2D, threads2D>>>(d_grayInput, d_resized, width, height, RESIZE_W, RESIZE_H);

    dctRowsKernel<<<RESIZE_H, RESIZE_W>>>(d_resized, d_dctTemp, RESIZE_W);
    dctColsKernel<<<RESIZE_W, RESIZE_H>>>(d_dctTemp, d_dctFinal, RESIZE_W);

    generateHashKernel<<<1, HASH_SIZE * HASH_SIZE>>>(d_dctFinal, d_hash);

    uint64_t hash;
    CHECK_CUDA(hipMemcpy(&hash, d_hash, sizeof(uint64_t), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_img));
    CHECK_CUDA(hipFree(d_grayInput));
    CHECK_CUDA(hipFree(d_resized));
    CHECK_CUDA(hipFree(d_dctTemp));
    CHECK_CUDA(hipFree(d_dctFinal));
    CHECK_CUDA(hipFree(d_hash));

    return hash;
}

int main(int argc, char** argv) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("SMs: %d\n", prop.multiProcessorCount);

    int img_count = 1000;

    vector<string> file_vec;
    try {
        int i = 0;
        for (const auto& entry : fs::directory_iterator(argv[1])) {
            i++;
            if (entry.is_regular_file() && entry.path().extension() == ".jpg")
                file_vec.push_back(entry.path());
            if (i == img_count) break;
        }
    } catch (const fs::filesystem_error& e) {
        cout << "Filesystem error: " << e.what() << endl;
        return 1;
    }
    cout << "Total images: " << file_vec.size() << endl;

    sort(file_vec.begin(), file_vec.end());


    const int batchSize = min(MAX_STREAMS, static_cast<int>(file_vec.size()));
    hipStream_t streams[MAX_STREAMS];
    for (int i = 0; i < batchSize; ++i)
        CHECK_CUDA(hipStreamCreate(&streams[i]));

    vector<uint64_t> hashes(file_vec.size());
    vector<unsigned char*> hostImgs(batchSize);
    vector<tuple<int, int, int>> imgShapes(batchSize);

    double duration = 0;
    for (size_t i = 0; i < file_vec.size(); i += batchSize) {
        int actualBatch = min((int)batchSize, (int)(file_vec.size() - i));

        for (int j = 0; j < actualBatch; ++j) {
            int width, height, channels;
            hostImgs[j] = stbi_load(file_vec[i + j].c_str(), &width, &height, &channels, 0);
            imgShapes[j] = {width, height, channels};
        }

        auto start = steady_clock::now();
        for (int j = 0; j < actualBatch; ++j) {
            auto [w, h, c] = imgShapes[j];
            computeHashAsync(hostImgs[j], w, h, c, streams[j], &hashes[i + j]);
        }

        auto end = steady_clock::now();
        double d = duration_cast<milliseconds>(end - start).count();
        duration += d;

        for (int j = 0; j < actualBatch; ++j) {
            CHECK_CUDA(hipStreamSynchronize(streams[j]));
            stbi_image_free(hostImgs[j]);
        }
    }

    for (int i = 0; i < batchSize; ++i)
        CHECK_CUDA(hipStreamDestroy(streams[i]));
    
    for (size_t i = 0; i < file_vec.size(); ++i) {
        string filename = fs::path(file_vec[i]).filename();
        cout << "Image: " << filename << endl;

        bitset<64> bitHash(hashes[i]);
        cout << "pHash: " << bitHash << endl;
    }
    
    printf("[CUDA Stream]:\t\t[%.3f] ms\n", duration);

    return 0;
}
