#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdint>
#include <cmath>
#include <bitset>
#include <string>
#include <filesystem>
#include "stb_image.h"

using namespace std;
using namespace chrono;
namespace fs = std::filesystem;


#define RESIZE_W 32
#define RESIZE_H 32
#define DCT_SIZE 32
#define HASH_SIZE 8

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__device__ float dct_coeff(int k, int N) {
    return (k == 0) ? sqrtf(1.0f / N) : sqrtf(2.0f / N);
}

__global__ void ucharToGrayscaleKernel(const unsigned char* input, float* output, int size, int channels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    if (channels == 3) {
        float r = input[i * 3 + 0];
        float g = input[i * 3 + 1];
        float b = input[i * 3 + 2];
        output[i] = 0.299f * r + 0.587f * g + 0.114f * b;
    } else {
        output[i] = input[i];
    }
}

__global__ void resizeBilinearKernel(const float* input, float* output, int inW, int inH, int outW, int outH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= outW || y >= outH) return;

    float scaleX = static_cast<float>(inW) / outW;
    float scaleY = static_cast<float>(inH) / outH;

    float srcX = (x + 0.5f) * scaleX - 0.5f;
    float srcY = (y + 0.5f) * scaleY - 0.5f;

    int x0 = max(0, min(inW - 2, static_cast<int>(floorf(srcX))));
    int y0 = max(0, min(inH - 2, static_cast<int>(floorf(srcY))));

    float dx = srcX - x0;
    float dy = srcY - y0;

    float val = 
        (1 - dx) * (1 - dy) * input[y0 * inW + x0] +
        dx * (1 - dy) * input[y0 * inW + (x0 + 1)] +
        (1 - dx) * dy * input[(y0 + 1) * inW + x0] +
        dx * dy * input[(y0 + 1) * inW + (x0 + 1)];

    output[y * outW + x] = val;
}

__global__ void dctRowsKernel(const float* input, float* output, int N) {
    int row = blockIdx.x;
    int u = threadIdx.x;

    if (row < N && u < N) {
        float sum = 0.0f;
        for (int x = 0; x < N; ++x) {
            sum += input[row * N + x] * cosf((M_PI * (2 * x + 1) * u) / (2.0f * N));
        }
        output[row * N + u] = dct_coeff(u, N) * sum;
    }
}

__global__ void dctColsKernel(const float* input, float* output, int N) {
    int col = blockIdx.x;
    int v = threadIdx.x;

    if (col < N && v < N) {
        float sum = 0.0f;
        for (int y = 0; y < N; ++y) {
            sum += input[y * N + col] * cosf((M_PI * (2 * y + 1) * v) / (2.0f * N));
        }
        output[v * N + col] = dct_coeff(v, N) * sum;
    }
}

__global__ void generateHashKernel(const float* dct, uint64_t* hashOut) {
    __shared__ float block[HASH_SIZE * HASH_SIZE];
    int idx = threadIdx.x;
    if (idx < HASH_SIZE * HASH_SIZE) {
        int i = idx / HASH_SIZE;
        int j = idx % HASH_SIZE;
        block[idx] = dct[i * DCT_SIZE + j];
    }
    __syncthreads();

    if (idx == 0) {
        float sum = 0;
        for (int i = 1; i < HASH_SIZE * HASH_SIZE; ++i) // skip DC
            sum += block[i];
        float avg = sum / (HASH_SIZE * HASH_SIZE - 1);

        uint64_t hash = 0;
        for (int i = 1; i < HASH_SIZE * HASH_SIZE; ++i)
            if (block[i] > avg)
                hash |= (1ULL << (i - 1));
        *hashOut = hash;
    }
}

uint64_t computeHashGPU(unsigned char* img, int width, int height, int channels) {
    int imgSize = width * height;

    float* d_grayInput;
    float* d_resized;
    float* d_dctTemp;
    float* d_dctFinal;
    uint64_t* d_hash;

    CHECK_CUDA(hipMalloc(&d_grayInput, imgSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_resized, RESIZE_W * RESIZE_H * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dctTemp, RESIZE_W * RESIZE_H * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dctFinal, RESIZE_W * RESIZE_H * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_hash, sizeof(uint64_t)));

    unsigned char* d_img;
    CHECK_CUDA(hipMalloc(&d_img, imgSize * channels * sizeof(unsigned char)));
    CHECK_CUDA(hipMemcpy(d_img, img, imgSize * channels * sizeof(unsigned char), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (imgSize + threads - 1) / threads;
    ucharToGrayscaleKernel<<<blocks, threads>>>(d_img, d_grayInput, imgSize, channels);

    dim3 threads2D(RESIZE_W, RESIZE_H);
    dim3 blocks2D(1, 1);
    resizeBilinearKernel<<<blocks2D, threads2D>>>(d_grayInput, d_resized, width, height, RESIZE_W, RESIZE_H);

    dctRowsKernel<<<RESIZE_H, RESIZE_W>>>(d_resized, d_dctTemp, RESIZE_W);
    dctColsKernel<<<RESIZE_W, RESIZE_H>>>(d_dctTemp, d_dctFinal, RESIZE_W);

    generateHashKernel<<<1, HASH_SIZE * HASH_SIZE>>>(d_dctFinal, d_hash);

    uint64_t hash;
    CHECK_CUDA(hipMemcpy(&hash, d_hash, sizeof(uint64_t), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_img));
    CHECK_CUDA(hipFree(d_grayInput));
    CHECK_CUDA(hipFree(d_resized));
    CHECK_CUDA(hipFree(d_dctTemp));
    CHECK_CUDA(hipFree(d_dctFinal));
    CHECK_CUDA(hipFree(d_hash));

    return hash;
}

int main(int argc, char** argv) {
    int img_count = 100;

    vector<string> file_vec;
    try {
        int i = 0;
        for (const auto& entry : fs::directory_iterator(argv[1])) {
            i++;
            if (entry.is_regular_file() && entry.path().extension() == ".jpg")
                file_vec.push_back(entry.path());
            if (i == img_count) break;
        }
    } catch (const fs::filesystem_error& e) {
        cout << "Filesystem error: " << e.what() << endl;
        return 1;
    }
    cout << "Total images: " << file_vec.size() << endl;

    sort(file_vec.begin(), file_vec.end());

    vector<pair<unsigned char*, tuple<int, int, int>>> img_vec;
    for (auto f: file_vec) {
        const char* file = f.c_str();
        int width, height, channels;
        unsigned char* img = stbi_load(file, &width, &height, &channels, 0);
        img_vec.push_back({img, {width, height, channels}});
    }

    vector<pair<string, string>> results;
    auto start = steady_clock::now();
    for (int i = 0; i < img_vec.size(); i++) {
        string filename = fs::path(file_vec[i]).filename();
        cout << "Image: " << filename << endl;

        auto img = img_vec[i].first;
        auto [width, height, channels] = img_vec[i].second;
        uint64_t hash = computeHashGPU(img, width, height, channels);
        stbi_image_free(img);

        bitset<64> bitHash(hash);
        cout << "pHash: " << bitHash << endl;

        results.push_back({filename, bitHash.to_string()});
    }
    

    auto end = steady_clock::now();   
    double duration = duration_cast<milliseconds>(end - start).count();
    printf("[CUDA V2]:\t\t[%.3f] ms\n", duration);

    return 0;
}
